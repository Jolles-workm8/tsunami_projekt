#include "hip/hip_runtime.h"
#include "cuda_WavePropagation2d.h"
#include "cuda_functions.cuh"
//#include "cuda_solver.cuh"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
using namespace cooperative_groups;


tsunami_lab::patches::cuda_WavePropagation2d::cuda_WavePropagation2d(
    t_idx i_xCells, t_idx i_yCells) {
  // set number of cells without GhostCells in x and y Direction
  m_xCells = i_xCells;
  m_yCells = i_yCells;

  size = (m_xCells + 2) * (m_yCells + 2);
  // allocate memory including ghostcells on each side
  m_h = (float *)malloc(size * sizeof(float));
  m_hu = (float *)malloc(size * sizeof(float));
  m_hv = (float *)malloc(size * sizeof(float));
  m_b = (float *)malloc(size * sizeof(float));

  // init to zero
  for (unsigned long l_ceY = 0; l_ceY < (m_yCells + 2); l_ceY++) {
    for (unsigned long l_ceX = 0; l_ceX < (m_xCells + 2); l_ceX++) {
      unsigned long l_ce = l_ceX + l_ceY * (m_xCells + 2);
      m_b[l_ce] = 0;
      m_h[l_ce] = 0;
      m_hu[l_ce] = 0;
      m_hv[l_ce] = 0;
    }
  }

  // allocate memory on GPU
  hipMalloc((void **)&h_dev, size * sizeof(float));
  hipMalloc((void **)&h_dev_UpdateL, size * sizeof(float));
  hipMalloc((void **)&h_dev_UpdateR, size * sizeof(float));
  hipMalloc((void **)&hu_dev, size * sizeof(float));
  hipMalloc((void **)&hu_dev_UpdateL, size * sizeof(float));
  hipMalloc((void **)&hu_dev_UpdateR, size * sizeof(float));
  hipMalloc((void **)&hv_dev, size * sizeof(float));
  hipMalloc((void **)&b_dev, size * sizeof(float));
}

tsunami_lab::patches::cuda_WavePropagation2d::~cuda_WavePropagation2d() {
  // free memory on CPU
  free(m_h);
  free(m_hu);
  free(m_hv);
  free(m_b);

  // free memory on GPU
  hipFree(h_dev);
  hipFree(h_dev_UpdateR);
  hipFree(h_dev_UpdateL);
  hipFree(hu_dev);
  hipFree(hu_dev_UpdateR);
  hipFree(hu_dev_UpdateL);
  hipFree(hv_dev);
  hipFree(b_dev);
}
void tsunami_lab::patches::cuda_WavePropagation2d::MemTransfer() {
  hipMemcpy(h_dev, m_h, size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(hu_dev, m_hu, size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(hv_dev, m_hv, size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(b_dev, m_b, size * sizeof(float), hipMemcpyHostToDevice);
}
void tsunami_lab::patches::cuda_WavePropagation2d::timeStep(
    t_real i_scaling, t_idx i_computeSteps) {
     printf("asfgh");
  t_idx l_nx = m_xCells + 2;
  t_idx l_ny = m_yCells + 2;

  /*
  dim3 threadsPerBlock(1,1);
  dim3 BlocksPerGrid(l_nx+1, l_ny+1);

  solverInit<<<BlocksPerGrid, threadsPerBlock>>>(
      h_dev, h_dev_UpdateR, h_dev_UpdateL, hu_dev, hu_dev_UpdateR, hu_dev_UpdateL, hv_dev,
      l_nx, l_ny, b_dev, i_computeSteps, i_scaling);
*/
int supportsCoopLaunch = 0;
hipDeviceGetAttribute(&supportsCoopLaunch, hipDeviceAttributeCooperativeLaunch, 0);
printf("test %d \n", supportsCoopLaunch);
if(supportsCoopLaunch){

  
  /// This will launch a grid that can maximally fill the GPU, on the default stream with kernel arguments
  int numBlocksPerSm = 0;
  // Number of threads my_kernel will be launched with
  int numThreads = 256;
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, solverInit, numThreads, 0);
  // launch
  void *kernelArgs[] = {h_dev, h_dev_UpdateR, h_dev_UpdateL, hu_dev, hu_dev_UpdateR, hu_dev_UpdateL, hv_dev,
      &l_nx, &l_ny, b_dev, &i_computeSteps, &i_scaling };
  dim3 dimBlock(16, 16, 1);
  dim3 dimGrid(deviceProp.multiProcessorCount*numBlocksPerSm, 1, 1);
  hipLaunchCooperativeKernel((void*)solverInit, dimGrid, dimBlock, kernelArgs);

  hipDeviceSynchronize();

  hipMemcpy(m_h, h_dev, size * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(m_hu, hu_dev, size * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(m_hv, hv_dev, size * sizeof(float), hipMemcpyDeviceToHost);
  
  } 
}

__global__ void solverInit(float *i_h, float *l_h_UpdateR, float *l_h_UpdateL, float *i_hu,
                           float *l_hu_UpdateR, float *l_hu_UpdateL, float *i_hv,
                           int i_nx, int i_ny, float *i_b, int i_iter,
                           float i_scaling) {
  
  printf("test \n");
  grid_group g = this_grid();
  printf("test %d\n", g.is_valid());
                            
  for (int i = 0; i < i_iter; ++i) {
    initUpdatesZero(l_h_UpdateR, l_h_UpdateR, l_hu_UpdateR, l_hu_UpdateL, i_nx, i_ny);
    setGhostOutflow(i_h, i_hu, i_hv, i_b, i_nx, i_ny);
    printf("test %d\n", g.is_valid());
    g.sync();
     printf("test123\n");

    //x Sweep
    
    netUpdates(i_h, l_h_UpdateR, l_h_UpdateR, i_hu, l_hu_UpdateR, l_hu_UpdateL, i_nx - 1, i_ny, i_b,
               i_scaling, i_nx, 1);
    g.sync();
    updateValues(i_h, l_h_UpdateR, l_h_UpdateR, i_hu, l_hu_UpdateR, l_hu_UpdateL, i_nx, i_ny);
    initUpdatesZero(l_h_UpdateR, l_h_UpdateR, l_hu_UpdateR, l_hu_UpdateL, i_nx, i_ny);
    g.sync();
  
    //y Sweep
    netUpdates(i_h, l_h_UpdateR, l_h_UpdateR, i_hv, l_hu_UpdateR, l_hu_UpdateL, i_nx, i_ny - 1, i_b,
               i_scaling, i_nx, i_nx);
    g.sync();
    updateValues(i_h, l_h_UpdateR, l_h_UpdateR, i_hv, l_hu_UpdateR, l_hu_UpdateL, i_nx, i_ny);
    g.sync();
  }
}

__device__ void updateValues(float *o_h, float *i_h_UpdateR, float *i_h_UpdateL, float *o_hu,
                           float *i_hu_UpdateR, float *i_hu_UpdateL, int i_nx, int i_ny){
  
  
  int l_i = blockIdx.x * blockDim.x + threadIdx.x;
  int l_j = blockIdx.y * blockDim.y + threadIdx.y;
  int idx = l_i + l_j * i_nx;

  if (l_i < i_nx && l_j < i_ny){
  o_h[idx] =o_h[idx] + i_h_UpdateR[idx] + i_h_UpdateL[idx];
  o_hu[idx] =o_hu[idx] + i_hu_UpdateR[idx] + i_hu_UpdateL[idx];      
  }                   
}

__device__ void initUpdatesZero(float *i_h_UpdateR, float *i_h_UpdateL, float *i_hu_UpdateR, float *i_hu_UpdateL, int i_nx, int i_ny){
  int l_i = blockIdx.x * blockDim.x + threadIdx.x;
  int l_j = blockIdx.y * blockDim.y + threadIdx.y;
  int idx = l_i + l_j * i_nx;

  if (l_i < i_nx && l_j < i_ny){
    i_h_UpdateR[idx] = 0;
    i_h_UpdateL[idx] = 0;
    i_hu_UpdateR[idx] = 0;
    i_hu_UpdateL[idx] = 0;
  }
}

__device__ void setGhostOutflow(float *i_height, float *i_hu, float *i_hv,
                                float *i_b, int i_nx, int i_ny) {
  int l_i = blockIdx.x * blockDim.x + threadIdx.x;
  int l_j = blockIdx.y * blockDim.y + threadIdx.y;
  int idx = l_i + l_j * i_nx;
  
  if (l_i < i_nx && l_j < i_ny){
    if (l_j == 0) {
      i_height[idx] = i_height[idx + i_nx];
      i_hu[idx] = i_hu[idx + i_nx];
      i_hv[idx] = i_hv[idx + i_nx];
      i_b[idx] = i_b[idx + i_nx];
    } else if (l_j == i_nx - 1) {
      i_height[idx] = i_height[idx - i_nx];
      i_hu[idx] = i_hu[idx - i_nx];
      i_hv[idx] = i_hv[idx - i_nx];
      i_b[idx] = i_b[idx - i_nx];
    }

    __syncthreads();

    if (l_i == 0) {
      i_height[idx] = i_height[idx + 1];
      i_hu[idx] = i_hu[idx + 1];
      i_hv[idx] = i_hv[idx + 1];
      i_b[idx] = i_b[idx + 1];
    } else if (l_i == i_ny - 1) {
      i_height[idx] = i_height[idx - 1];
      i_hu[idx] = i_hu[idx - 1];
      i_hv[idx] = i_hv[idx - 1];
      i_b[idx] = i_b[idx - 1];
    }
  }
}

__device__ void netUpdates(float *i_height, float *o_height_UpdateR, float *o_height_UpdateL,
                           float *i_momentum, float *o_momentum_UpdateR, float *o_momentum_UpdateL,
                           int i_xEdges, int i_yEdges, float *i_b, float i_scaling,
                           int i_nx, int i_stride) {
  
  printf("Edges %d and %d\n",i_xEdges, i_yEdges);
  int l_i = blockIdx.x * blockDim.x + threadIdx.x;
  int l_j = blockIdx.y * blockDim.y + threadIdx.y;
  int idx = l_i + l_j * i_nx;

  float m_gSqrt = sqrtf(9.812);
  float m_g = 9.812;

  if (l_i < i_xEdges && l_j < i_yEdges) {
    printf("Hello from block %d, thread %d with access to %d and %d and %d and %d \n", l_i, l_j, idx, idx+i_stride, i_xEdges, i_yEdges);
    // compute u for left and right
    float l_uL = i_momentum[idx] / i_height[idx];
    float l_uR = i_momentum[idx + i_stride] / i_height[idx + i_stride];

    float l_hL = i_height[idx];
    float l_hR = i_height[idx + i_stride];

    float l_huL = i_momentum[idx];
    float l_huR = i_momentum[idx + i_stride];

    float l_bL = i_b[idx];
    float l_bR = i_b[idx + i_stride];


    if(l_bL >= 0){
      l_hL = l_hR;
      l_huL = l_huR;
      l_bL = l_bR;
    }
    else if(l_bR >= 0){
      l_hR = l_hL;
      l_huR = l_huL;
      l_bR = l_bL;
    }

    // compute WaveSpeed ,

    float l_hSqrtL = sqrtf(l_hL);
    float l_hSqrtR = sqrtf(l_hR);

    float l_hRoe = 0.5f * (l_hL + l_hR);
    float l_uRoe = l_hSqrtL * l_uL + l_hSqrtR * l_uR;
    l_uRoe /= l_hSqrtL + l_hSqrtR;

    float l_ghSqrtRoe = m_gSqrt * sqrtf(l_hRoe);


    float l_waveSpeedL = l_uRoe - l_ghSqrtRoe;
    float l_waveSpeedR = l_uRoe + l_ghSqrtRoe;

    float l_detInv = 1 / (l_waveSpeedR - l_waveSpeedL);

    // compute the bathymetry effect
    float l_bathEff = -m_g * (l_bR - l_bL) * (l_hL + l_hR) / 2;

    // compute jump in the flux
    float l_fJump_1 = l_huR - l_huL;
    float l_fJump_2 = l_huR * l_huR / l_hR - l_huL * l_huL / l_hL +
                      (m_g / 2) * (l_hR * l_hR - l_hL * l_hL);
    l_fJump_2 -= l_bathEff;

    // compute the alpha values
    float l_strengthL =
        -i_scaling * l_detInv * (l_waveSpeedR * l_fJump_1 - l_fJump_2);
    float l_strengthR =
        -i_scaling * l_detInv * (l_fJump_2 - l_waveSpeedL * l_fJump_1);


    if (l_waveSpeedL < 0) {
      o_height_UpdateL[idx] = l_strengthL;
      o_momentum_UpdateL[idx] = l_strengthL * l_waveSpeedL;
    } else {
      o_height_UpdateR[idx + i_stride] = l_strengthL;
      o_momentum_UpdateR[idx + i_stride] = l_strengthL * l_waveSpeedL;
    }

    if (l_waveSpeedR > 0) {
      o_height_UpdateR[idx + i_stride] = l_strengthR;
      o_momentum_UpdateR[idx + i_stride] = l_strengthR * l_waveSpeedR;

    } else {
      o_height_UpdateL[idx] = l_strengthR;
      o_momentum_UpdateL[idx] = l_strengthR * l_waveSpeedR;
    }

    if(i_b[idx] >= 0){
    o_height_UpdateL[idx] = 0;
    o_momentum_UpdateL[idx] = 0;
    }
    if(i_b[idx + i_stride] >= 0){
    o_height_UpdateR[idx + i_stride] = 0;
    o_momentum_UpdateR[idx + i_stride] = 0;
    }
  }
}

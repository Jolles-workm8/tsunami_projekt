#include <hip/hip_runtime.h>

#include <iostream>

__device__ void netUpdates(float *i_height_old, float *i_height_new,
                           float *i_momentum_old, float *i_momentum_new,
                           int i_nx, int i_ny, float *i_b, float i_scaling) {
  int scaling = 4;
  int l_i = blockIdx.x * blockDim.x + threadIdx.x;
  int l_j = blockIdx.y * blockDim.y + threadIdx.y;
  int idx = l_i * scaling + l_j;

  int i_displ = 1;
  float m_gSqrt = sqrtf(9.812);
  float m_g = 9.812;

  if (l_i < i_nx - 1 && l_j < i_ny - 1) {
    // compute u for left and right
    float l_uL = i_momentum_old[idx] / i_height_old[idx];
    float l_uR = i_momentum_old[idx + i_displ] / i_height_old[idx + i_displ];

    float l_hL = i_height_old[idx];
    float l_hR = i_height_old[idx + i_displ];

    float l_huL = i_momentum_old[idx];
    float l_huR = i_momentum_old[idx + i_displ];

    float l_bL = i_b[idx];
    float l_bR = i_b[idx + i_displ];

    // compute WaveSpeed ,

    float l_hSqrtL = sqrtf(l_hL);
    float l_hSqrtR = sqrtf(l_hR);

    float l_hRoe = 0.5f * (l_hL + l_hR);
    float l_uRoe = l_hSqrtL * l_uL + l_hSqrtR * l_uR;
    l_uRoe /= l_hSqrtL + l_hSqrtR;

    float l_ghSqrtRoe = m_gSqrt * sqrtf(l_hRoe);

    float l_waveSpeedL = l_uRoe - l_ghSqrtRoe;
    float l_waveSpeedR = l_uRoe + l_ghSqrtRoe;

    float l_detInv = 1 / (l_waveSpeedR - l_waveSpeedL);

    // compute the bathymetry effect
    float l_bathEff = -m_g * (l_bR - l_bL) * (l_hL + l_hR) / 2;

    // compute jump in the flux
    float l_fJump_1 = l_huR - l_huL;
    float l_fJump_2 = l_huR * l_huR / l_hR - l_huL * l_huL / l_hL +
                      (m_g / 2) * (l_hR * l_hR - l_hL * l_hL);
    l_fJump_2 -= l_bathEff;

    // compute the alpha values
    float l_strengthL =
        -i_scaling * l_detInv * (l_waveSpeedR * l_fJump_1 - l_fJump_2);
    float l_strengthR =
        -i_scaling * l_detInv * (l_fJump_2 - l_waveSpeedL * l_fJump_1);

    __syncthreads();

    if (l_waveSpeedL < 0) {
      atomicAdd(&i_height_new[idx], l_strengthL);
      atomicAdd(&i_momentum_new[idx], l_strengthL * l_waveSpeedL);
    } else {
      atomicAdd(&i_height_new[idx + i_displ], l_strengthL);
      atomicAdd(&i_momentum_new[idx + i_displ], l_strengthL * l_waveSpeedL);
    }

    if (l_waveSpeedR > 0) {
      atomicAdd(&i_height_new[idx + i_displ], l_strengthR);
      atomicAdd(&i_momentum_new[idx + i_displ], l_strengthR * l_waveSpeedR);
    } else {
      atomicAdd(&i_height_new[idx], l_strengthR);
      atomicAdd(&i_momentum_new[idx], l_strengthR * l_waveSpeedR);
    }
    __syncthreads();

    i_height_old[idx] = i_height_new[idx];
    i_momentum_old[idx] = i_momentum_new[idx];
  }
}

__global__ void WavePropagation2d(float *i_h_old, float *i_h_new,
                                  float *i_hu_old, float *i_hu_new,
                                  float *i_hv_new, float *i_hv_old, int i_nx,
                                  int i_ny, float *i_b, int i_iter,
                                  float i_scaling) {
  for (int i = 0; i < i_iter; ++i) {
    netUpdates(i_h_old, i_h_new, i_hu_old, i_hu_new, i_nx, i_ny, i_b,
               i_scaling);
    __syncthreads();
  }
}

int main() {
  int N = 128;
  int iteration = 32;
  int size = 16;
  int i_nx = 4;
  int i_ny = 4;
  float scaling = 0.5;
  float *h_host, *hu_host, *hv_host, *h_dev_new, *h_dev_old, *hu_dev_new,
      *hu_dev_old, *hv_dev_new, *hv_dev_old, *b_host, *b_dev;
  h_host = (float *)malloc(size * sizeof(float));
  hu_host = (float *)malloc(size * sizeof(float));
  hu_host = (float *)malloc(size * sizeof(float));
  b_host = (float *)malloc(size * sizeof(float));
  hipMalloc((void **)&h_dev_old, size * sizeof(float));
  hipMalloc((void **)&h_dev_new, size * sizeof(float));
  hipMalloc((void **)&hu_dev_old, size * sizeof(float));
  hipMalloc((void **)&hu_dev_new, size * sizeof(float));
  hipMalloc((void **)&hv_dev_old, size * sizeof(float));
  hipMalloc((void **)&hv_dev_new, size * sizeof(float));
  hipMalloc((void **)&b_dev, size * sizeof(float));

  for (int i = 0; i < size; i++) {
    h_host[i] = i;
    hu_host[i] = i;
    hv_host[i] = i;
    b_host[i] = 0;
  }

  hipMemcpy(h_dev_old, h_host, size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(hu_dev_old, hu_host, size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(hv_dev_old, hv_host, size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(b_dev, b_host, size * sizeof(float), hipMemcpyHostToDevice);

  hipError_t err = hipGetLastError();
  dim3 threadsPerBlock(i_nx, i_ny);
  for (int i = 0; i < N / iteration; i++) {
    WavePropagation2d<<<1, threadsPerBlock>>>(
        h_dev_old, h_dev_new, hu_dev_old, hu_dev_new, hv_dev_new, hv_dev_old,
        i_nx, i_ny, b_dev, iteration, scaling);

    hipDeviceSynchronize();
  }
  hipMemcpy(h_host, h_dev_old, size * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(hu_host, hu_dev_old, size * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(hv_host, hv_dev_old, size * sizeof(float), hipMemcpyDeviceToHost);

  //
  // for (int i = 0; i < size; i++) {
  //   std::cout << h_host[i] << '\n';
  // }

  free(h_host);
  free(hu_host);
  free(b_host);

  hipFree(h_dev_new);
  hipFree(h_dev_old);
  hipFree(hu_dev_new);
  hipFree(hu_dev_old);
  hipFree(b_dev);

  return 0;
}

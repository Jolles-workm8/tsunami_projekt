#include <hip/hip_runtime.h>

#include <iostream>

__device__ void netUpdates(float *i_height_old, float *i_height_new,
                           float *i_momentum_old, float *i_momentum_new,
                           int i_nx, int i_ny, float *i_b, float i_scaling,
                           int idx, int i_stride) {
  int l_i = blockIdx.x * blockDim.x + threadIdx.x;
  int l_j = blockIdx.y * blockDim.y + threadIdx.y;

  float m_gSqrt = sqrtf(9.812);
  float m_g = 9.812;

  i_height_new[idx] = i_height_old[idx];
  i_momentum_new[idx] = i_momentum_old[idx];

  if (l_i < i_nx && l_j < i_ny) {
    // compute u for left and right
    float l_uL = i_momentum_old[idx] / i_height_old[idx];
    float l_uR = i_momentum_old[idx + i_stride] / i_height_old[idx + i_stride];

    float l_hL = i_height_old[idx];
    float l_hR = i_height_old[idx + i_stride];

    float l_huL = i_momentum_old[idx];
    float l_huR = i_momentum_old[idx + i_stride];

    float l_bL = i_b[idx];
    float l_bR = i_b[idx + i_stride];

    // compute WaveSpeed ,

    float l_hSqrtL = sqrtf(l_hL);
    float l_hSqrtR = sqrtf(l_hR);

    float l_hRoe = 0.5f * (l_hL + l_hR);
    float l_uRoe = l_hSqrtL * l_uL + l_hSqrtR * l_uR;
    l_uRoe /= l_hSqrtL + l_hSqrtR;

    float l_ghSqrtRoe = m_gSqrt * sqrtf(l_hRoe);

    float l_waveSpeedL = l_uRoe - l_ghSqrtRoe;
    float l_waveSpeedR = l_uRoe + l_ghSqrtRoe;

    float l_detInv = 1 / (l_waveSpeedR - l_waveSpeedL);

    // compute the bathymetry effect
    float l_bathEff = -m_g * (l_bR - l_bL) * (l_hL + l_hR) / 2;

    // compute jump in the flux
    float l_fJump_1 = l_huR - l_huL;
    float l_fJump_2 = l_huR * l_huR / l_hR - l_huL * l_huL / l_hL +
                      (m_g / 2) * (l_hR * l_hR - l_hL * l_hL);
    l_fJump_2 -= l_bathEff;

    // compute the alpha values
    float l_strengthL =
        -i_scaling * l_detInv * (l_waveSpeedR * l_fJump_1 - l_fJump_2);
    float l_strengthR =
        -i_scaling * l_detInv * (l_fJump_2 - l_waveSpeedL * l_fJump_1);

    __syncthreads();

    if (l_waveSpeedL < 0) {
      atomicAdd(&i_height_new[idx], l_strengthL);
      atomicAdd(&i_momentum_new[idx], l_strengthL * l_waveSpeedL);
    } else {
      atomicAdd(&i_height_new[idx + i_stride], l_strengthL);
      atomicAdd(&i_momentum_new[idx + i_stride], l_strengthL * l_waveSpeedL);
    }

    if (l_waveSpeedR > 0) {
      atomicAdd(&i_height_new[idx + i_stride], l_strengthR);
      atomicAdd(&i_momentum_new[idx + i_stride], l_strengthR * l_waveSpeedR);
    } else {
      atomicAdd(&i_height_new[idx], l_strengthR);
      atomicAdd(&i_momentum_new[idx], l_strengthR * l_waveSpeedR);
    }
    __syncthreads();
  }
  i_height_old[idx] = i_height_new[idx];
  i_momentum_old[idx] = i_momentum_new[idx];
}

__device__ void setGhostOutflow(float *i_height, float *i_hu, float *i_hv,
                                float *i_b, int i_nx, int i_ny) {
  int l_i = blockIdx.x * blockDim.x + threadIdx.x;
  int l_j = blockIdx.y * blockDim.y + threadIdx.y;
  int idx = l_i + l_j * i_nx;

  if (l_j == 0) {
    i_height[idx] = i_height[idx + i_nx];
    i_hu[idx] = i_hu[idx + i_nx];
    i_hv[idx] = i_hv[idx + i_nx];
    i_b[idx] = i_b[idx + i_nx];
  } else if (l_j == i_nx - 1) {
    i_height[idx] = i_height[idx - i_nx];
    i_hu[idx] = i_hu[idx - i_nx];
    i_hv[idx] = i_hv[idx - i_nx];
    i_b[idx] = i_b[idx - i_nx];
  }

  __syncthreads();

  if (l_i == 0) {
    i_height[idx] = i_height[idx + 1];
    i_hu[idx] = i_hu[idx + 1];
    i_hv[idx] = i_hv[idx + 1];
    i_b[idx] = i_b[idx + 1];
  } else if (l_i == i_ny - 1) {
    i_height[idx] = i_height[idx - 1];
    i_hu[idx] = i_hu[idx - 1];
    i_hv[idx] = i_hv[idx - 1];
    i_b[idx] = i_b[idx - 1];
  }
}

__global__ void WavePropagation2d(float *i_h_old, float *i_h_new,
                                  float *i_hu_old, float *i_hu_new,
                                  float *i_hv_old, float *i_hv_new, int i_nx,
                                  int i_ny, float *i_b, int i_iter,
                                  float i_scaling) {
  int l_i = blockIdx.x * blockDim.x + threadIdx.x;
  int l_j = blockIdx.y * blockDim.y + threadIdx.y;
  int idx = l_i + l_j * i_nx;
  for (int i = 0; i < i_iter; ++i) {
    setGhostOutflow(i_h_old, i_hu_old, i_hv_old, i_b, i_nx, i_ny);
    __syncthreads();

    netUpdates(i_h_old, i_h_new, i_hu_old, i_hu_new, i_nx - 1, i_ny, i_b,
               i_scaling, idx, 1);
    __syncthreads();

    netUpdates(i_h_old, i_h_new, i_hv_old, i_hv_new, i_nx, i_ny - 1, i_b,
               i_scaling, idx, i_nx);
    __syncthreads();
  }
  //printf("%f, %d\n", i_h_old[idx], idx);
}

void cudaWaveProp() {
  int N = 1;
  int iteration = 1;
  int size = 64;
  int i_nx = std::sqrt(size);
  int i_ny = std::sqrt(size);
  float scaling = 0.001;
  float *h_host, *hu_host, *hv_host, *h_dev_new, *h_dev_old, *hu_dev_new,
      *hu_dev_old, *hv_dev_new, *hv_dev_old, *b_host, *b_dev;
  h_host = (float *)malloc(size * sizeof(float));
  hu_host = (float *)malloc(size * sizeof(float));
  hv_host = (float *)malloc(size * sizeof(float));
  b_host = (float *)malloc(size * sizeof(float));
  hipMalloc((void **)&h_dev_old, size * sizeof(float));
  hipMalloc((void **)&h_dev_new, size * sizeof(float));
  hipMalloc((void **)&hu_dev_old, size * sizeof(float));
  hipMalloc((void **)&hu_dev_new, size * sizeof(float));
  hipMalloc((void **)&hv_dev_old, size * sizeof(float));
  hipMalloc((void **)&hv_dev_new, size * sizeof(float));
  hipMalloc((void **)&b_dev, size * sizeof(float));

  for (int i = 0; i < size; i++) {
    h_host[i] = i + 1;
    hu_host[i] = 1;
    hv_host[i] = 1;
    b_host[i] = 1;
  }

  hipMemcpy(h_dev_old, h_host, size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(hu_dev_old, hu_host, size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(hv_dev_old, hv_host, size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(b_dev, b_host, size * sizeof(float), hipMemcpyHostToDevice);

  hipError_t err = hipGetLastError();
  dim3 threadsPerBlock(4, 4);
  dim3 BlocksPerGrid(i_nx / 4, i_ny / 4);
  for (int i = 0; i < N / iteration; i++) {
    WavePropagation2d<<<BlocksPerGrid, threadsPerBlock>>>(
        h_dev_old, h_dev_new, hu_dev_old, hu_dev_new, hv_dev_old, hv_dev_new,
        i_nx, i_ny, b_dev, iteration, scaling);

    hipDeviceSynchronize();
  }
  hipMemcpy(h_host, h_dev_old, size * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(hu_host, hu_dev_old, size * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(hv_host, hv_dev_old, size * sizeof(float), hipMemcpyDeviceToHost);
  // std::cout << h_host[4] << '\n';
  //
  // for (int i = 0; i < size; i++) {
  //   std::cout << h_host[i] << '\n';
  // }

  free(h_host);
  free(hu_host);
  free(b_host);

  hipFree(h_dev_new);
  hipFree(h_dev_old);
  hipFree(hu_dev_new);
  hipFree(hu_dev_old);
  hipFree(hv_dev_new);
  hipFree(hv_dev_old);
  hipFree(b_dev);

}

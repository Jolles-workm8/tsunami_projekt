#include <hip/hip_runtime.h>

#include <iostream>

__global__ void netUpdates(float *i_h, float *i_hu, int i_nx, int i_ny,
                           float *i_b) {
  int scaling = 4;
  int l_i = blockIdx.x * blockDim.x + threadIdx.x;
  int l_j = blockIdx.y * blockDim.y + threadIdx.y;
  int tid = threadIdx.x + threadIdx.y * blockDim.x;
  int idx = l_i * scaling + l_j;

  int i_displ = 1;
  float m_gSqrt = sqrtf(9.812);
  float m_g = 9.812;

  // new values after on iteration
  extern __shared__ float height[];
  extern __shared__ float momentum[];

  // height[idx] = 0;
  // momentum[idx] = 0;

  if (l_i < i_nx && l_j < i_ny) {
    printf("%d, %d\n", l_i, l_j);

    // compute u for left and right
    float l_uL = i_hu[idx] / i_h[idx];
    float l_uR = i_hu[idx + i_displ] / i_h[idx + i_displ];

    float l_hL = i_h[idx];
    float l_hR = i_h[idx + i_displ];

    float l_huL = i_hu[idx];
    float l_huR = i_hu[idx + i_displ];

    float l_bL = i_b[idx];
    float l_bR = i_b[idx + i_displ];

    __syncthreads();

    // compute WaveSpeed ,

    float l_hSqrtL = sqrtf(l_hL);
    float l_hSqrtR = sqrtf(l_hR);

    float l_hRoe = 0.5f * (l_hL + l_hR);
    float l_uRoe = l_hSqrtL * l_uL + l_hSqrtR * l_uR;
    l_uRoe /= l_hSqrtL + l_hSqrtR;

    float l_ghSqrtRoe = m_gSqrt * sqrtf(l_hRoe);

    __syncthreads();

    float l_waveSpeedL = l_uRoe - l_ghSqrtRoe;
    float l_waveSpeedR = l_uRoe + l_ghSqrtRoe;

    float l_detInv = 1 / (l_waveSpeedR - l_waveSpeedL);

    // compute the bathymetry effect
    float l_bathEff = -m_g * (l_bR - l_bL) * (l_hL + l_hR) / 2;

    // compute jump in the flux
    float l_fJump_1 = l_huR - l_huL;
    float l_fJump_2 = l_huR * l_huR / l_hR - l_huL * l_huL / l_hL +
                      (m_g / 2) * (l_hR * l_hR - l_hL * l_hL);
    l_fJump_2 -= l_bathEff;

    // compute the alpha values
    float l_strengthL = l_detInv * (l_waveSpeedR * l_fJump_1 - l_fJump_2);
    float l_strengthR = l_detInv * (l_fJump_2 - l_waveSpeedL * l_fJump_1);

    printf("Hello world  %f\n", l_strengthL);
    __syncthreads();

    if (l_waveSpeedL < 0) {
      atomicAdd(&height[idx], l_strengthL);
      atomicAdd(&momentum[idx], l_strengthL * l_waveSpeedL);
    } else {
      atomicAdd(&height[idx + i_displ], l_strengthL);
      atomicAdd(&momentum[idx + i_displ], l_strengthL * l_waveSpeedL);
    }

    if (l_waveSpeedR > 0) {
      atomicAdd(&height[idx + i_displ], l_strengthR);
      atomicAdd(&momentum[idx + i_displ], l_strengthR * l_waveSpeedR);
    } else {
      atomicAdd(&height[idx], l_strengthR);
      atomicAdd(&momentum[idx], l_strengthR * l_waveSpeedR);
    }

    __syncthreads();
  }
}

int main() {
  int size = 16;
  int i_nx = 4;
  int i_ny = 4;
  float *h_host, *hu_host, *h_dev, *hu_dev, *b_host, *b_dev;
  h_host = (float *)malloc(size * sizeof(float));
  hu_host = (float *)malloc(size * sizeof(float));
  b_host = (float *)malloc(size * sizeof(float));
  hipMalloc((void **)&h_dev, size * sizeof(float));
  hipMalloc((void **)&hu_dev, size * sizeof(float));
  hipMalloc((void **)&b_dev, size * sizeof(float));

  for (int i = 0; i < size; i++) {
    h_host[i] = i;
    hu_host[i] = i;
    b_host[i] = 0;
  }

  hipMemcpy(h_dev, h_host, size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(hu_dev, hu_host, size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(b_dev, b_host, size * sizeof(float), hipMemcpyHostToDevice);

  hipError_t err = hipGetLastError();
  dim3 threadsPerBlock(4, 4);

  netUpdates<<<1, threadsPerBlock>>>(h_dev, hu_dev, i_nx, i_ny, b_dev);

  hipMemcpy(h_host, h_dev, size * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(hu_host, hu_dev, size * sizeof(float), hipMemcpyDeviceToHost);
  //
  // for (int i = 0; i < size; i++) {
  //   std::cout << h_host[i] << '\n';
  // }

  free(h_host);
  free(hu_host);
  free(b_host);

  hipFree(h_dev);
  hipFree(hu_dev);
  hipFree(b_dev);

  return 0;
}
